
#include <hip/hip_runtime.h>
#include <iostream> 
#include <hipblas.h>
#include <time.h>

#define HEIGHT 1024
#define WIDTH 1024
#define BLOCK_SIZE 32 //this now refers to abstacted block size 
#define kernal_size 3

using namespace std; 


__global__ void matrix_mult(int *a, int *b, int *c){
	int threadRow = threadIdx.y;
	int threadCol = threadIdx.x;


	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;


		__shared__ int a_share[BLOCK_SIZE + kernal_size][BLOCK_SIZE + kernal_size];
		__shared__ int b_share[kernal_size][kernal_size];

		// each thread reads one element from A matrix 
		//think about over indexing a here  
		int get_row = blockDim.y * (HEIGHT/BLOCK_SIZE) + threadIdx.y;
		int get_col = blockDim.x * (WIDTH/BLOCK_SIZE) + threadIdx.x; 
		
		//a_share[threadRow][threadCol] = a[get_row][get_col];
		if (get_row<HEIGHT && get_col<WIDTH ) 
		{
			if (threadRow<(HEIGHT/BLOCK_SIZE) && threadCol<(WIDTH/BLOCK_SIZE))
			{
				a_share[threadRow][threadCol] = a[get_row*WIDTH + get_col];
		    }
		    __syncthreads();
			if (!(threadRow<(HEIGHT/BLOCK_SIZE) && threadCol<(WIDTH/BLOCK_SIZE)))
			{
				a_share[threadRow][threadCol] = a[get_row*WIDTH + get_col];
		    }
			
		}
		//the first kernal_size^2 threads will read in b 
		if (threadRow<kernal_size && threadCol<kernal_size)
		{
			b_share[threadRow][threadCol] = b[threadRow*kernal_size + threadCol];
		}
		// make sure the sub-matrices are loaded before starting the computation
		__syncthreads();

		if (threadRow<BLOCK_SIZE && threadCol<BLOCK_SIZE && get_row< (HEIGHT-kernal_size) 
			&& get_col <(WIDTH-kernal_size))
		{
			for (int i = 0; i<kernal_size; i++)
			{
				
				for (int j = 0; j<kernal_size; j++)
				{
					//c[get_row][get_col]+= b_share[i][j] * a_share[threadRow+i][threadRow+j];
					c[get_row*(WIDTH-kernal_size) + get_col]+= b_share[i][j] * a_share[threadRow+i][threadRow+j];
					
					//c[get_row*(WIDTH-kernal_size) + get_col]+= b_share[i][j];

				}
			}
		}
		// make sure every thread is done computing before loading new sub-matrices
		__syncthreads();

	

	
}

int main(){
	
    int i;
    //int **a = (int**)malloc(sizeof(*int) * HEIGHT);
    //for (int i = 0; i < HEIGHT; i++) a[i] = malloc(sizeof(a[i]) * WIDTH);
    int *a = (int*)malloc(sizeof(int) * HEIGHT * WIDTH);
    //int** a = new *int[HEIGHT];
    //for (int i = 0; i < HEIGHT; i++) a[i] = new int[WIDTH];
	//int** b = new *int[kernal_size];
	//for (int i = 0; i < kernal_size; i++) b[i] = new int[kernal_size];
    //int **b = (int**)malloc(sizeof(*int) * kernal_size);
    //for (int i = 0; i < kernal_size; i++) b[i] = malloc(sizeof(b[i]) * kernal_size);
    int *b = (int*)malloc(sizeof(int) * kernal_size * kernal_size);
	int new_height = HEIGHT - kernal_size +1;
	int new_width = WIDTH - kernal_size +1;
	int *c = (int*)malloc(sizeof(int) * new_height * new_width);
    //int **c = new *int[new_height];
    //for (int i = 0; i < new_height; i++) c[i] = new int[new_width];
    //int **c = (int**)malloc(sizeof(*int) * new_height);
    //for (int i = 0; i < new_height; i++) c[i] = malloc(sizeof(c[i]) * new_width);


	for(int i=0; i<WIDTH; i++)
	{
		for(int j = 0; j<HEIGHT; j++)
		{
			//a[i][j]=1;
			a[i*WIDTH + j] = 1; 
		}
	}
	for (int i = 0; i<kernal_size; i++)
	{
		for(int j =0; j < kernal_size; j++)
		{
			b[i*kernal_size + j] = 1; 
		}
	}

	
  	




	int *gpu_a, *gpu_b, *gpu_c;
	hipMalloc((void**)&gpu_a, sizeof(int) * HEIGHT * WIDTH);
	hipMalloc((void**)&gpu_b, sizeof(int) * kernal_size * kernal_size);
	hipMalloc((void**)&gpu_c, sizeof(int) * new_height * new_width);

	struct timespec start, stop;
	double time;

	hipMemcpy(gpu_a, a, sizeof(int) * HEIGHT * WIDTH, hipMemcpyHostToDevice);
	hipMemcpy(gpu_b, b, sizeof(int) * kernal_size * kernal_size, hipMemcpyHostToDevice);

	 

	dim3 dimGrid(32, 32);
	dim3 dimBlock(32+kernal_size, 32+kernal_size);

	//if( clock_gettime( CLOCK_REALTIME, &start) == -1 ) { perror( "clock gettime" );}
	cout<<"test"<<endl;
	matrix_mult<<<dimGrid, dimBlock>>>(gpu_a, gpu_b, gpu_c);
	hipMemcpy(c, gpu_c, sizeof(int) * new_width * new_height, hipMemcpyDeviceToHost);

	//if( clock_gettime( CLOCK_REALTIME, &stop) == -1 ) { perror( "clock gettime" );}
	//time = (stop.tv_sec - start.tv_sec)+ (double)(stop.tv_nsec - start.tv_nsec)/1e9;
	//printf("time is %f ns\n", time*1e9);

	//printf("c[451][451]=%d\n", c[451*1024+451]);

	
	cout<<"c[451][451]="<<c[451*1024+451];

	free(a);
	free(b);
	free(c);
	hipFree(gpu_a);
	hipFree(gpu_b);
	hipFree(gpu_c);
	return 0;
}
