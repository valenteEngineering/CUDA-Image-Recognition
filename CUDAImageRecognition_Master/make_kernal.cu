#include "hip/hip_runtime.h"
#include <iostream>
#include <cmath>
#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>
#include <opencv2/imgproc/imgproc.hpp>

using namespace cv;
using namespace std;

int   kernal_size = 3;
int stdev = kernal_size / 3;

const double pi = 3.14159;
double exp_arg, sum_y, norm_factor;

int main(int argc, char* argv[])
{
	int mean = 0;

	// open image 
	if( argc != 2)
    {
     cout <<" Usage: display_image ImageToLoadAndDisplay" << endl;
     return -1;
    }

    Mat image;
    image = imread(argv[1], CV_LOAD_IMAGE_COLOR);   // Read the file

    if(! image.data )                              // Check for invalid input
    {
        cout <<  "Could not open or find the image" << std::endl ;
        return -1;
    }

	int lower_bound = mean - (kernal_size / 2);
	int upper_bound = mean + (kernal_size / 2);
	int* x = new int [kernal_size];
	double* y = new double [kernal_size];
	double** kernal = new double* [kernal_size];
	for (int i = 0; i < kernal_size; i++) {kernal[i] = new double[kernal_size];}
	double k = (1 / (pow(2 * pi, .5) * stdev));

	for (int i = 0; i < (kernal_size); i++) {
		x[i] = lower_bound + i; 
	}

	for (int i = 0; i < kernal_size; i++)
	{
		exp_arg = ( pow(((double)x[i] - mean), 2));
		exp_arg = -(exp_arg / (2 * pow(stdev, 2)));
		y[i] = (k * exp(exp_arg));
		sum_y += y[i];
	}

	norm_factor = 1 / sum_y;

	for (int i = 0; i < kernal_size; i++) {
		y[i] = (y[i] * norm_factor); 
	}

	for (int i = 0; i < kernal_size; i++)
	{

		for (int j = 0; j < kernal_size; j++)
		{
			kernal[i][j] = y[i] * y[j];
		}
	}

	//must use periodic or whatever type of edge padding
	// we may split up matrix for convolution but not into
	//blocks which are smaller than the kernal itself

	Mat outputImage = image.clone();
	outputImage.setTo(cv::Scalar(0,0,0));

	int output_height = image.size().height - kernal_size + 1;
	int output_width = image.size().width - kernal_size + 1;

	for (int i = 0 ; i < output_height ; i++)
	{
		for (int j = 0 ; j < output_width; j++)
		{
			for (int k = 0; k < 3; k++)
			{
				for (int h = i ; h < i + kernal_size ; h++)
				{
					for (int w = j ; w < j + kernal_size ; w++)
					{
						outputImage.at<Vec3b>(i,j)[k] += kernal[h - i][w - j] * image.at<Vec3b>(h,w)[k];
					}
				}
			}	
		}
	}

	namedWindow( "Display window", WINDOW_AUTOSIZE );     // Create a window for display.
    imshow( "Display window", outputImage );             // Show our image inside it.

    waitKey(0);                                          // Wait for a keystroke in the window

	// free memory 
	delete x;
	delete y;
	for (int i = 0; i < kernal_size; i++) {
		delete kernal[i];
	}
	delete kernal;

	return 0;
}