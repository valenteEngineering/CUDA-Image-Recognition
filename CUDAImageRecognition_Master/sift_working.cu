#include "hip/hip_runtime.h"
#include <iostream>
#include <cmath>
#include <vector>
#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>
#include <opencv2/imgproc/imgproc.hpp>

using namespace cv;
using namespace std;

const double pi = 3.14159;

double** make_kernel(int kernel_size, double stdev) {
	double exp_arg, sum_y, norm_factor;
	int mean = 0;
	int lower_bound = mean - (kernel_size / 2);
	int* x = new int [kernel_size];
	double* y = new double [kernel_size];
	double** kernel = new double* [kernel_size];
	for (int i = 0; i < kernel_size; i++) {kernel[i] = new double[kernel_size];}
	double k = (1 / (pow(2 * pi, .5) * stdev));

	for (int i = 0; i < (kernel_size); i++) {
		x[i] = lower_bound + i; 
	}

	for (int i = 0; i < kernel_size; i++)
	{
		exp_arg = ( pow(((double)x[i] - mean), 2));
		exp_arg = -(exp_arg / (2 * pow(stdev, 2)));
		y[i] = (k * exp(exp_arg));
		sum_y += y[i];
	}

	norm_factor = 1 / sum_y;

	for (int i = 0; i < kernel_size; i++) {
		y[i] = (y[i] * norm_factor); 
	}

	for (int i = 0; i < kernel_size; i++)
	{

		for (int j = 0; j < kernel_size; j++)
		{
			kernel[i][j] = y[i] * y[j];
		}
	}
	delete x;
	delete y;
	return kernel; 
}

void free_kernel_mem(double** kernel, int kernel_size) {
	for (int i = 0; i < kernel_size; i++) {
		delete kernel[i];
	}
	delete kernel;
}

void showImage(Mat image) {
	namedWindow("Display window", WINDOW_AUTOSIZE);     
    imshow("Display window", image);             
    waitKey(0);                                          
}

int main(int argc, char* argv[])
{
	// open image 
	if( argc != 2)
    {
     cout <<" Usage: display_image ImageToLoadAndDisplay" << endl;
     return -1;
    }

    Mat image;
    image = imread(argv[1], CV_LOAD_IMAGE_COLOR);   // Read the file

    if(! image.data )                              // Check for invalid input
    {
        cout <<  "Could not open or find the image" << std::endl ;
        return -1;
    }

    vector<Mat> octaves;
    vector<Mat> LoGs;
	int num_octaves = 5;

	// create octaves with Gaussian Blur 
	for (int octave = 0; octave < num_octaves; octave++) 
	{
		Mat outputImage = image.clone();
		outputImage.setTo(cv::Scalar(0,0,0));
		int kernel_size = 3*pow(sqrt(2),octave);
		int output_height = image.size().height;
		int output_width = image.size().width;
		double** kernel = make_kernel(kernel_size, (double)kernel_size/3.0);
		int x, y;
		for (int i = 0 ; i < output_height; i++)
		{
			for (int j = 0 ; j < output_width; j++)
			{
				for (int k = 0; k < 3; k++)
				{
					for (int h = i - (kernel_size/2); h <= i + (kernel_size/2); h++)
					{
						for (int w = j - (kernel_size/2); w <= j + (kernel_size/2); w++)
						{
							x=h;
							y=w;
							if (h < 0) x=-h;
							else if (h>=output_height) {x=(2*output_height - h -1);}
							if (w < 0) y = -w;
							else if (w>=output_width) {y = (2*output_width - w -1); }
							outputImage.at<Vec3b>(i,j)[k] += 
							kernel[h-i+(kernel_size/2)][w-j+(kernel_size/2)] * image.at<Vec3b>(x,y)[k];
						}
					}
				}	
			}
		}

	for (int i = 0; i < octaves.size(); i++) {
		showImage(octaves[i]);
	}

	for (int curr_octave = 1; curr_octave < num_octaves; curr_octave++) {
		Mat LoG = image.clone();
		for (int i = 0; i < image.size().height; i++) {
			for (int j = 0; j < image.size().width; j++) {
				for (int k = 0; k < 3; k++) {
					LoG.at<Vec3b>(i,j)[k] = abs(
						octaves[curr_octave].at<Vec3b>(i,j)[k] - 
						octaves[curr_octave - 1].at<Vec3b>(i,j)[k]);
				}
			}
		}
		LoGs.push_back(LoG);
	}

	for (int i = 0; i < LoGs.size(); i++) {
		showImage(LoGs[i]);
	}

	for(int curr_octave = 1; curr_octave < 3; curr_octave++) {
		for (int i = 0; i < image.size().height; i++) {
			for (int j = 0; j < image.size().width; j++) {
				bool isGreatest = true;
				bool isLeast = true;
				double d1 = pow(LoGs[curr_octave].at<Vec3b>(i,j)[0],2) + pow(LoGs[curr_octave].at<Vec3b>(i,j)[1],2) + pow(LoGs[curr_octave].at<Vec3b>(i,j)[2],2);
				for (int level = -1; level < 2; level++) {
					if (!isGreatest && !isLeast) {
						break;
					}
					for (int x = -1; x < 2; x++) {
						if (!isGreatest && !isLeast) {
							break;
						}
						for (int y = -1; y < 2; y++) {
							double d2 = pow(LoGs[curr_octave + level].at<Vec3b>(i+x,j+y)[0],2) + 
								 pow(LoGs[curr_octave + level].at<Vec3b>(i+x,j+y)[1],2) + 
								 pow(LoGs[curr_octave + level].at<Vec3b>(i+x,j+y)[2],2);
							if (i+x >= 0 && 
								i+x <  image.size().height &&
								j+y >= 0 &&                 
								j+y <  image.size().width &&
								d1 > d2) 
							{
								isLeast = false;
							}
							if (i+x >= 0 && 
								i+x < image.size().height &&
								j+y >= 0 && 
								j+y < image.size().width &&
								d1 < d2)
							{
								isGreatest = false;
							}
							if (!isGreatest && !isLeast) {
								break;
							} else if (level == 1 && x == 1 && y == 1) {
								image.at<Vec3b>(i,j)[0] = 0;
								image.at<Vec3b>(i,j)[1] = 0;
								image.at<Vec3b>(i,j)[2] = 255;
							}
						}
					}
				}
			}
		}
	}

	showImage(image);

	return 0;
}